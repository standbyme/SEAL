#include "hip/hip_runtime.h"
#include "iostream"
#include "main.cuh"
__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

__host__ void haha(uint32_t a, uint32_t b)
{
    cuda_hello<<<1,1>>>();
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
    }
}